﻿
#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <ctime>

__global__ void tabuSearchKernel(int* weights, int* values, int* solutions, int capacity, int numItems, int* tabuList, int tabuListSize, int maxIterations) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < numItems) {
        // Inicjalizacja najlepszego rozwiązania i rozwiązania lokalnego
        int bestSolutionValue = 0;
        int bestSolutionWeight = 0;
        int localSolutionValue = 0;
        int localSolutionWeight = 0;

        // Inicjalizacja listy tabu
        for (int i = 0; i < tabuListSize; i++) {
            tabuList[i] = 0;
        }

        for (int iter = 0; iter < maxIterations; iter++) {
            int itemToAdd = -1;
            int bestDelta = 0;

            // Przejrzyj wszystkie przedmioty i wybierz najlepszy do dodania/usunięcia
            for (int i = 0; i < numItems; i++) {
                int deltaValue = values[i] - localSolutionValue;
                int deltaWeight = weights[i] - localSolutionWeight;

                // Sprawdź, czy przedmiot może być dodany i nie jest na liście tabu
                if (deltaWeight + localSolutionWeight <= capacity && tabuList[i] < iter) {
                    if (deltaValue > bestDelta) {
                        bestDelta = deltaValue;
                        itemToAdd = i;
                    }
                }
            }

            // Aktualizacja rozwiązania lokalnego i najlepszego
            if (itemToAdd != -1) {
                localSolutionValue += values[itemToAdd];
                localSolutionWeight += weights[itemToAdd];
                tabuList[itemToAdd] = iter + tabuListSize; // Aktualizacja listy tabu

                if (localSolutionValue > bestSolutionValue) {
                    bestSolutionValue = localSolutionValue;
                    bestSolutionWeight = localSolutionWeight;
                    solutions[index] = itemToAdd; // Zapis najlepszego rozwiązania
                }
            }
        }
    }
}

int main() {
    const int numItems = 10000; // Liczba elementów
    const int capacity = numItems;  // Pojemność plecaka
    const int tabuListSize = 10; // Rozmiar listy tabu
    const int maxIterations = 1000; // Maksymalna liczba iteracji
    const int blockSizes[] = { 64, 128, 256, 512, 1024 }; // Rozmiary bloków do testowania
    const int numTests = sizeof(blockSizes) / sizeof(blockSizes[0]);

    int weights[numItems]; // Wagi przedmiotów
    int values[numItems];  // Wartości przedmiotów
    int solutions[numItems]; // Rozwiązania

    // Zmienne do przechowywania czasu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int N = 50; // Maksymalna wartość dla wagi i wartości

    // Inicjalizacja generatora liczb losowych
    srand(time(0));

    // Inicjalizacja wag i wartości przedmiotów
    for (int i = 0; i < numItems; i++) {
        weights[i] = rand() % (N + 1); // Losowa waga od 0 do N
        values[i] = rand() % (N + 1);  // Losowa wartość od 0 do N
    }

    int* d_weights, * d_values, * d_solutions, * d_tabuList;

    // Alokacja pamięci na GPU
    hipMalloc(&d_weights, numItems * sizeof(int));
    hipMalloc(&d_values, numItems * sizeof(int));
    hipMalloc(&d_solutions, numItems * sizeof(int));
    hipMalloc(&d_tabuList, tabuListSize * sizeof(int));

    // Kopiowanie danych na GPU
    hipMemcpy(d_weights, weights, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, numItems * sizeof(int), hipMemcpyHostToDevice);

    // Uruchomienie kernela
    for (int i = 0; i < numTests; ++i) {
        dim3 blockSize = blockSizes[i]; // Rozmiar bloku
        dim3 gridSize((numItems + blockSize.x - 1) / blockSize.x); // Rozmiar siatki

        // Zmienne do przechowywania czasu
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Start pomiaru czasu
        hipEventRecord(start);
        tabuSearchKernel <<<gridSize, blockSize>>> (d_weights, d_values, d_solutions, capacity, numItems, d_tabuList, tabuListSize, maxIterations);

        // Przeniesienie wyników do pamięci CPU i obliczenia
        int cpuBestCost = 0;
        hipMemcpy(solutions, d_solutions, numItems * sizeof(int), hipMemcpyDeviceToHost);
        // Koniec pomiaru czasu
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        for (int i = 0; i < numItems; i++) {
            if (solutions[i] != 0) {
                cpuBestCost += values[i]; // Sumowanie wartości przedmiotów w najlepszym rozwiązaniu
            }
        }

        std::cout << "Block size: " << blockSizes[i] << ", Best cost found: " << cpuBestCost << ", Execution time: " << milliseconds << " ms" << std::endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Sprzątanie
    hipFree(d_weights);
    hipFree(d_values);
    hipFree(d_solutions);
    hipFree(d_tabuList);

    return 0;
}